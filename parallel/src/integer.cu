#include "hip/hip_runtime.h"
#include "integer.h"

#include <string.h>
#include <stdio.h>
#include <stdlib.h>

__device__ void gcd(volatile uint32_t *x, volatile uint32_t *y);
__device__ void shiftR1(volatile uint32_t *x);
__device__ void shiftL1(volatile uint32_t *x);
__device__ int geq(volatile uint32_t *x, volatile uint32_t *y);
__device__ void cuSubtract(volatile uint32_t *x, volatile uint32_t *y, volatile uint32_t *z);

/* kernel */
__global__ void cuda_factorKeys(const integer *keys, uint32_t *notCoprime, size_t pitch, int tileRow, int tileCol, int tileDim, int numKeys) {
  __shared__ volatile uint32_t x[BLOCK_DIM][BLOCK_DIM][32];
  __shared__ volatile uint32_t y[BLOCK_DIM][BLOCK_DIM][32];

  int keyX = tileCol * tileDim + blockIdx.x * BLOCK_DIM + threadIdx.y;
  int keyY = tileRow * tileDim + blockIdx.y * BLOCK_DIM + threadIdx.z;

  if (keyX < numKeys && keyY < numKeys && keyX < keyY) {
    x[threadIdx.y][threadIdx.z][threadIdx.x] = keys[keyX].ints[threadIdx.x];
    y[threadIdx.y][threadIdx.z][threadIdx.x] = keys[keyY].ints[threadIdx.x];

    gcd(x[threadIdx.y][threadIdx.z], y[threadIdx.y][threadIdx.z]);

    if (threadIdx.x == 31) {
      y[threadIdx.y][threadIdx.z][threadIdx.x] -= 1;

      if (__any(y[threadIdx.y][threadIdx.z][threadIdx.x])) {
        /* int notCoprimeKeyX = keyX - tileCol * tileDim; */
        /* int notCoprimeKeyY = keyY - tileRow * tileDim; */

        /* uint32_t *notCoprimeRow = (uint32_t *) ((char *) notCoprime + notCoprimeKeyX * pitch); */
        /* int notCoprimeCol = notCoprimeKeyY / 32; */
        /* int notCoprimeOffset = notCoprimeKeyY % 32; */

        /* notCoprimeRow[notCoprimeCol] |= 1 << notCoprimeOffset; */
      }
    }
  }
}

void cuda_wrapper(dim3 gridDim, dim3 blockDim, integer* d_keys, uint32_t* d_notCoprime,
    size_t pitch, int tileRow, int tileCol, int tileDim, int numKeys) {
      cuda_factorKeys<<<gridDim, blockDim>>>(d_keys, d_notCoprime,
          pitch, tileRow, tileCol, tileDim, numKeys);
}

__device__ void gcd(volatile uint32_t *x, volatile uint32_t *y) {
  int tid = threadIdx.x;

  while (__any(x[tid])) {
    while ((x[31] & 1) == 0)
      shiftR1(x);

    while ((y[31] & 1) == 0)
      shiftR1(y);

    if (geq(x, y)) {
      cuSubtract(x, y, x);
      shiftR1(x);
    }
    else {
      cuSubtract(y, x, y);
      shiftR1(y);
    }
  }
}

__device__ void shiftR1(volatile uint32_t *x) {
  int tid = threadIdx.x;
  uint32_t prevX = tid ? x[tid-1] : 0;
  x[tid] = (x[tid] >> 1) | (prevX << 31);
}

__device__ void shiftL1(volatile uint32_t *x) {
  int tid = threadIdx.x;
  uint32_t nextX = tid != 31 ? x[tid+1] : 0;
  x[tid] = (x[tid] << 1) | (nextX >> 31);
}

__device__ int geq(volatile uint32_t *x, volatile uint32_t *y) {
  __shared__ unsigned int pos[BLOCK_DIM][BLOCK_DIM];
  int tid = threadIdx.x;

  if (tid == 0)
    pos[threadIdx.y][threadIdx.z] = 31;

  if (x[tid] != y[tid])
    atomicMin(&pos[threadIdx.y][threadIdx.z], tid);

  return x[pos[threadIdx.y][threadIdx.z]] >= y[pos[threadIdx.y][threadIdx.z]];
}

__device__ void cuSubtract(volatile uint32_t *x, volatile uint32_t *y, volatile uint32_t *z) {
  __shared__ unsigned char s_borrow[BLOCK_DIM][BLOCK_DIM][32];
  unsigned char *borrow = s_borrow[threadIdx.y][threadIdx.z];
  int tid = threadIdx.x;

  if (tid == 0)
    borrow[31] = 0;

  uint32_t t;
  t = x[tid] - y[tid];

  if(tid)
    borrow[tid - 1] = (t > x[tid]);

  while (__any(borrow[tid])) {
    if (borrow[tid])
      t--;

    if (tid)
      borrow[tid - 1] = (t == 0xFFFFFFFFu && borrow[tid]);
  }

  z[tid] = t;
}
